
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <math.h>
namespace cuda
{
  namespace elementwise
  {
    // An empirical parameter
    // In the mainstream GPU architecture, the maximum number of registers per block is 64K,
    // the maximum number of registers that can be used by each thread is 255.
    // So, kThreadsPerBlock = 64 * 1024 / 255 = 256.
    // Refer from https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities
    constexpr uint kThreadsPerBlock = 256;
    // An empirical parameter
    constexpr uint kWaves = 32;
    constexpr uint kStride = 2;

    struct CudaConfig
    {
      int dev_{0};
      int sm_nums_{1};
      int max_threads_{1};
    };

    // Get some necessary hardware config.
    inline hipError_t GetCurrentConfig(CudaConfig *config)
    {
      // 1. Get current device.
      // 2. Get current sm_nums
      // 3. Get the maximum resident threads in per multiprocessor.
      int dev;
      hipError_t err = hipGetDevice(&dev);
      if (err != hipSuccess)
      {
        return err;
      }
      int sm_nums;
      err = hipDeviceGetAttribute(&sm_nums, hipDeviceAttributeMultiprocessorCount, dev);
      if (err != hipSuccess)
      {
        return err;
      }
      int max_threads;
      err = hipDeviceGetAttribute(&max_threads, hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);
      if (err != hipSuccess)
      {
        return err;
      }
      config->dev_ = dev;
      config->sm_nums_ = sm_nums;
      config->max_threads_ = max_threads;
      return err;
    }

    // Get best blocks basing on parallel data size for current hardware, adaptively.
    inline uint GetBestBlocks(uint n, const CudaConfig &config)
    {
      uint best_blocks =
          std::max<uint>(1, std::min<uint>((n + kThreadsPerBlock - 1) / kThreadsPerBlock,
                                           config.sm_nums_ * config.max_threads_ / kThreadsPerBlock * kWaves));
      return best_blocks;
    }

    template <typename T, uint vec_size>
    struct VectorizedTraitType
    {
      using type = typename std::aligned_storage<vec_size * sizeof(T), vec_size * sizeof(T)>::type;
    };

    template <typename T, uint vec_size>
    using VectorizedType = typename VectorizedTraitType<T, vec_size>::type;

    template <typename T, uint VecSize>
    union Vec
    {
      static_assert(sizeof(VectorizedType<T, VecSize>) == sizeof(T) * VecSize, "data can not be aligned.");
      __device__ Vec() {}
      VectorizedType<T, VecSize> storage_;
      T elements_[VecSize];
    };

    template <typename T, uint VecSize>
    struct alignas(sizeof(T) * VecSize) AlignVec
    {
      T elements_[VecSize];
    };

    constexpr uint kMaxVecBytes = 128 / 8;
    constexpr uint kMaxVecSize = 8;

    constexpr uint MsMin(uint a, uint b) { return a < b ? a : b; }

    template <typename T>
    constexpr uint VecSize()
    {
      return MsMin(kMaxVecBytes / sizeof(T), kMaxVecSize);
    }

    template <typename T, typename U, typename... Args>
    constexpr uint VecSize()
    {
      return MsMin(VecSize<T>(), VecSize<U, Args...>());
    }

    template <typename T>
    class CheckApply2
    {
      typedef char apply_unit;
      struct apply_struct
      {
        char x_[2];
      };

      template <typename IN3>
      static apply_unit check(decltype(&IN3::Apply2));
      template <typename IN3>
      static apply_struct check(...);

    public:
      enum
      {
        value = sizeof(check<T>(0)) == sizeof(char)
      };
    };

    template <uint vec_size>
    bool IsAligned()
    {
      return true;
    }

    template <uint vec_size, typename T, typename... Args>
    bool IsAligned(const T *ptr, const Args *...others)
    {
      return reinterpret_cast<uintptr_t>(ptr) % sizeof(Vec<T, vec_size>) == 0 && IsAligned<vec_size, Args...>(others...);
    }

    template <uint vec_size, typename FunctorT, typename OUT, typename... IN>
    __device__ typename std::enable_if<CheckApply2<FunctorT>::value == true && vec_size % kStride == 0,
                                       AlignVec<OUT, vec_size>>::type
    ApplyVec(const FunctorT &functor, const IN... in[vec_size])
    {
      AlignVec<OUT, vec_size> ret;

#pragma unroll
      for (uint j = 0; j < vec_size; j += kStride)
      {
        functor.Apply2(ret.elements_ + j, (in + j)...);
      }
      return ret;
    }

    template <uint vec_size, typename FunctorT, typename OUT, typename... IN>
    __device__ typename std::enable_if<CheckApply2<FunctorT>::value == false || vec_size % kStride != 0,
                                       AlignVec<OUT, vec_size>>::type
    ApplyVec(const FunctorT &functor, const IN... in[vec_size])
    {
      AlignVec<OUT, vec_size> ret;
#pragma unroll
      for (uint j = 0; j < vec_size; ++j)
      {
        ret.elements_[j] = functor((in[j])...);
      }
      return ret;
    }

    template <uint vec_size, bool tail, typename Factory, typename OUT, typename... IN>
    __global__ void __launch_bounds__(kThreadsPerBlock)
        DoApply(Factory factory, uint vec_nums, AlignVec<OUT, vec_size> *vec_out, const AlignVec<IN, vec_size> *...vec_in,
                uint tail_nums, OUT *tail_out, const IN *...tail_in)
    {
      auto functor = factory();
      const uint global_tid = blockIdx.x * kThreadsPerBlock + threadIdx.x;
      for (uint i = global_tid; i < vec_nums; i += blockDim.x * gridDim.x)
      {
        vec_out[i] = ApplyVec<vec_size, decltype(functor), OUT, IN...>(functor, (vec_in[i].elements_)...);
      }
      if (tail && global_tid < tail_nums)
      {
        tail_out[global_tid] = functor((tail_in[global_tid])...);
      }
    }

    template <uint vec_size, typename Factory, typename OUT, typename... IN>
    hipError_t LaunchKernel(Factory factory, uint nums, OUT *out, const IN *...in, hipStream_t stream)
    {
      const uint vec_nums = nums / vec_size;
      const uint tail_offset = vec_nums * vec_size;
      const uint tail_nums = nums - tail_offset;
      CudaConfig config;
      hipError_t err = GetCurrentConfig(&config);
      if (err != hipSuccess)
      {
        return err;
      }
      uint num_blocks = GetBestBlocks(vec_nums, config);
      dim3 block{kThreadsPerBlock};
      dim3 grid{uint(num_blocks)};
      if (tail_nums > 0)
      {
        auto func = DoApply<vec_size, true, Factory, OUT, IN...>;
        func<<<grid, block, 0, stream>>>(factory, vec_nums, reinterpret_cast<AlignVec<OUT, vec_size> *>(out),
                                         (reinterpret_cast<const AlignVec<IN, vec_size> *>(in))..., tail_nums,
                                         out + tail_offset, (in + tail_offset)...);
      }
      else
      {
        auto func = DoApply<vec_size, false, Factory, OUT, IN...>;
        func<<<grid, block, 0, stream>>>(factory, vec_nums, reinterpret_cast<AlignVec<OUT, vec_size> *>(out),
                                         (reinterpret_cast<const AlignVec<IN, vec_size> *>(in))..., tail_nums,
                                         out + tail_offset, (in + tail_offset)...);
      }
      return hipPeekAtLastError();
    }

    template <typename Factory, typename OUT, typename... IN>
    struct DoLaunch
    {
      static hipError_t Launch(Factory factory, uint n, OUT *out, const IN *...in, hipStream_t stream)
      {
        constexpr uint max_pack_size = VecSize<OUT, IN...>();
        if (IsAligned<max_pack_size, OUT, IN...>(out, in...))
        {
          return LaunchKernel<max_pack_size, Factory, OUT, IN...>(factory, n, out, in..., stream);
        }
        return LaunchKernel<1, Factory, OUT, IN...>(factory, n, out, in..., stream);
      }
    };

    template <typename FunctorT>
    struct TransitFactory
    {
      explicit TransitFactory(FunctorT functor) : transit_impl_(functor) {}
      __device__ FunctorT operator()() const { return transit_impl_; }

    private:
      FunctorT transit_impl_;
    };

    // API elementwise for input: a, output: out.
    template <typename Factory, typename OUT, typename IN>
    inline hipError_t UnaryTransit(Factory factory, uint n, OUT *out, const IN *in, hipStream_t stream)
    {
      return DoLaunch<Factory, OUT, IN>::Launch(factory, n, out, in, stream);
    }

    template <typename FunctorT, typename OUT, typename IN>
    inline hipError_t Unary(FunctorT functor, uint n, OUT *out, const IN *in, hipStream_t stream)
    {
      return UnaryTransit(TransitFactory<FunctorT>(functor), n, out, in, stream);
    }

    template <typename Factory, typename OUT, typename IN, typename IN2>
    inline hipError_t BinaryTransit(Factory factory, uint n, OUT *out, const IN *in, const IN2 *in2, hipStream_t stream)
    {
      return DoLaunch<Factory, OUT, IN, IN2>::Launch(factory, n, out, in, in2, stream);
    }

    // API elementwise for input: [a, b], output: out.
    template <typename FunctorT, typename OUT, typename IN, typename IN2>
    inline hipError_t Binary(FunctorT functor, uint n, OUT *out, const IN *in, const IN2 *in2, hipStream_t stream)
    {
      return BinaryTransit(TransitFactory<FunctorT>(functor), n, out, in, in2, stream);
    }

    template <typename Factory, typename OUT, typename IN, typename IN2, typename IN3>
    inline hipError_t TernaryTransit(Factory factory, uint n, OUT *out, const IN *in, const IN2 *in2, const IN3 *in3,
                                      hipStream_t stream)
    {
      return DoLaunch<Factory, OUT, IN, IN2, IN3>::Launch(factory, n, out, in, in2, in3, stream);
    }

    // API elementwise for input: [a, b, c], output: out.
    template <typename FunctorT, typename OUT, typename IN, typename IN2, typename IN3>
    inline hipError_t Ternary(FunctorT functor, uint n, OUT *out, const IN *in, const IN2 *in2, const IN3 *in3,
                               hipStream_t stream)
    {
      return TernaryTransit(TransitFactory<FunctorT>(functor), n, out, in, in2, in3, stream);
    }

    template <typename Factory, typename OUT, typename IN, typename IN2, typename IN3, typename IN4, typename IN5, typename IN6, typename IN7, typename IN8>
    inline hipError_t EightInputsTransit(Factory factory, uint n, OUT *out, const IN *in, const IN2 *in2, const IN3 *in3, const IN4 *in4, const IN5 *in5, const IN6 *in6, const IN7 *in7, const IN8 *in8,
                                          hipStream_t stream)
    {
      return DoLaunch<Factory, OUT, IN, IN2, IN3, IN4, IN5, IN6, IN7, IN8>::Launch(factory, n, out, in, in2, in3, in4, in5, in6, in7, in8, stream);
    }

    template <typename FunctorT, typename OUT, typename IN, typename IN2, typename IN3, typename IN4, typename IN5, typename IN6, typename IN7, typename IN8>
    inline hipError_t EightInputs(FunctorT functor, uint n, OUT *out, const IN *in, const IN2 *in2, const IN3 *in3, const IN4 *in4, const IN5 *in5, const IN6 *in6, const IN7 *in7, const IN8 *in8,
                                   hipStream_t stream)
    {
      return EightInputsTransit(TransitFactory<FunctorT>(functor), n, out, in, in2, in3, in4, in5, in6, in7, in8, stream);
    }

    template <typename Factory, typename OUT, typename IN, typename IN2, typename IN3, typename IN4>
    inline hipError_t FourInputsTransit(Factory factory, uint n, OUT *out, const IN *in, const IN2 *in2, const IN3 *in3, const IN4 *in4,
                                         hipStream_t stream)
    {
      return DoLaunch<Factory, OUT, IN, IN2, IN3, IN4>::Launch(factory, n, out, in, in2, in3, in4, stream);
    }

    template <typename FunctorT, typename OUT, typename IN, typename IN2, typename IN3, typename IN4>
    inline hipError_t FourInputs(FunctorT functor, uint n, OUT *out, const IN *in, const IN2 *in2, const IN3 *in3, const IN4 *in4,
                                  hipStream_t stream)
    {
      return FourInputsTransit(TransitFactory<FunctorT>(functor), n, out, in, in2, in3, in4, stream);
    }

    template <typename Factory, typename OUT, typename IN, typename IN2, typename IN3, typename IN4, typename IN5>
    inline hipError_t FiveInputsTransit(Factory factory, uint n, OUT *out, const IN *in, const IN2 *in2, const IN3 *in3, const IN4 *in4, const IN5 *in5,
                                         hipStream_t stream)
    {
      return DoLaunch<Factory, OUT, IN, IN2, IN3, IN4, IN5>::Launch(factory, n, out, in, in2, in3, in4, in5, stream);
    }

    template <typename FunctorT, typename OUT, typename IN, typename IN2, typename IN3, typename IN4, typename IN5>
    inline hipError_t FiveInputs(FunctorT functor, uint n, OUT *out, const IN *in, const IN2 *in2, const IN3 *in3, const IN4 *in4, const IN5 *in5,
                                  hipStream_t stream)
    {
      return FiveInputsTransit(TransitFactory<FunctorT>(functor), n, out, in, in2, in3, in4, in5, stream);
    }
  } // namespace elementwise
} // namespace cuda